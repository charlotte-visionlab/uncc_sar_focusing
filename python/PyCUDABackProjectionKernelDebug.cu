#include "hip/hip_runtime.h"
/* vim: set tabstop=4, shiftwidth=4, expandtab */
#include "mex.h"    /* Matlab junk */

#include <stdio.h>  /* printf */
#include "hip/hip_runtime.h"   /* CUDA */
#include <cutil_inline.h>
#include <cutil.h>
#include <time.h>

//#include "PyCUDABackProjectionKernel.h" /* #defines and prototypes and such */



/***
 * Compiler logics
 * **/
#  define MY_CUDA_SAFE_CALL_NO_SYNC( call) {                                    \
    hipError_t err = call;                                                    \
    if( hipSuccess != err) {                                                \
        printf( "Cuda error in file '%s' in line %i : %s.\n",                \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
    } }

#  define MY_CUDA_SAFE_CALL( call)     MY_CUDA_SAFE_CALL_NO_SYNC(call);             \



#define CLIGHT 299792458.0f /* c: speed of light, m/s */
#define PI 3.14159265359f   /* pi, accurate to 6th place in single precision */
#define PI2 6.2831853071800f   /* 2*pi */
#define PI_4__CLIGHT (4.0f * PI / CLIGHT)

#define REAL(vec) (vec.x)
#define IMAG(vec) (vec.y)

#define CAREFUL_AMINUSB_SQ(x,y) __fmul_rn(__fadd_rn((x), -1.0f*(y)), __fadd_rn((x), -1.0f*(y)))

#define ASSUME_Z_0    1        /* Ignore consult_DEM() and assume height = 0. */
#define USE_FAST_MATH 0     /* Use __math() functions? */
#define USE_RSQRT     0

#define MEXDEBUG      1

#define FLOAT_CLASS   mxSINGLE_CLASS

#ifndef VERBOSE
#define VERBOSE       0
#endif

#define BLOCKWIDTH    16
#define BLOCKHEIGHT   16

#define ZEROCOPY      0

#define MAKERADIUS(xpixel,ypixel, xa,ya,za) sqrtf(CAREFUL_AMINUSB_SQ(xpixel, xa) + CAREFUL_AMINUSB_SQ(ypixel, ya) + __fmul_rn(za, za))

/* Pound defines from my PyCUDA implementation:
 * 
 * ---Physical constants---
 * CLIGHT
 * PI
 *
 * ---Radar/data-specific constants---
 * Delta-frequency
 * Number of projections
 *
 * ---Application runtime constants---
 * Nfft, projection length
 * Image dimensions in pixels
 * Top-left image corner
 * X/Y pixel spacing
 *
 * ---Complicated constants---
 * PI_4_F0__CLIGHT = 4*pi/clight * radar_start_frequency
 * C__4_DELTA_FREQ = clight / (4 * radar_delta_frequency)
 * R_START_PRE = C__4_DELTA_FREQ * Nfft / (Nfft-1)
 *
 * ---CUDA constants---
 * Block dimensions
 */



/***
 * Type defs
 * ***/
typedef float FloatType; /* FIXME: this should be used everywhere */

/* From ATK imager */
typedef struct {
    float * real;
    float * imag;
} complex_split;

/* To work seamlessly with Hartley's codebase */
typedef complex_split bp_complex_split;


/***
 * Prototypes
 * ***/

float2 * format_complex_to_columns(bp_complex_split a, int width_orig, 
        int height_orig);

float2 * format_complex(bp_complex_split a, int size);

float4 * format_x_y_z_r(float * x, float * y, float * z, float * r, int size);

void run_bp(bp_complex_split phd, 
        float * xObs, float * yObs, float * zObs, float * r,
        int my_num_phi, int my_proj_length, int nxout, int nyout, 
        int image_chunk_width, int image_chunk_height, 
        int device, 
        bp_complex_split host_output_image,
        int start_output_index, int num_output_rows,
        float c__4_delta_freq, float pi_4_f0__clight, 
        float * start_frequencies,
        float left, float right, float bottom, float top,
        float min_eff_idx, float total_proj_length);

void convert_f0(float * vec, int N);
float extract_f0(float * vec, int N);

__global__ void backprojection_loop(float2 * full_image, 
        int Nphi, int nyout,
        float delta_pixel_x, float delta_pixel_y, float R_START_PRE, 
        int PROJ_LENGTH, int X_OFFSET, int Y_OFFSET,
        float C__4_DELTA_FREQ, float * PI_4_F0__CLIGHT, float left, float bottom,
        float min_eff_idx, float4 * platform_info,
        float * debug_effective_idx, float * debug_2, float * x_mat, float * y_mat,
        float rmin, float rmax);

__device__ float2 expjf(float in);
__device__ float2 expjf_div_2(float in);



/*
void testing_backprojection_loop(float2 * full_image,
        int Nphi, int nyout, float delta_pixel_x, float delta_pixel_y, 
        float R_START_PRE, int PROJ_LENGTH,
        int X_OFFSET, int Y_OFFSET,
        float C__4_DELTA_FREQ, float PI_4_F0__CLIGHT, 
        float left, float bottom, 
        int blockIdxx, int blockIdxy, int threadidxx, int threadidxy, 
        float * output_idxs);
        */




/* Globals and externs */

/* Complex textures containing range profiles */
texture<float2, 2, hipReadModeElementType> tex_projections;   

/* 4-elem. textures for x, y, z, r0 */
texture<float4, 1, hipReadModeElementType> tex_platform_info; 

void convert_f0(float * vec, int N) {
    int i;
    for (i=0; i<N; ++i)
        vec[i] *= PI_4__CLIGHT;
}

float extract_f0(float * vec, int N) {
    /* Mean ...
    int i;
    float sum = 0;
    for (i=0; i<N; ++i) {
        sum += vec[i];
    }
    return sum / N;
    */
    return vec[0];
}


/* 
 * Application parameters:
 *  - range profiles
 *
 * 
 * ATK imager gets the following:
 * - range profiles (complex)
 * - f0, vector of start frequencies, Hz
 * - r0, vector of distances from radar to center of illuminated scene, m
 * - x, y, z, vectors of radar position (x points east, y north, z up), m
 * - Nimgx, Nimgy, number of pixels in x and y
 * - deltaf, spacing of frequency vector, Hz
 * - Left, right, top, bottom, corners of the square on the ground to image
 */
void mexFunction(int nlhs,     /* number of LHS (output) arguments */
        mxArray *plhs[],       /* array of mxArray pointers to outputs */
        int nrhs,              /* number of RHS (input) args */
        const mxArray *prhs[]) /* array of pointers to inputs*/
{
    /* Section 1. 
     * These are the variables we'll use */
    /* Subsection A: these come from Matlab and are the same as the ATK code */
    complex_split range_profiles;
    float * start_frequencies;
    float * aimpoint_ranges;
    float * xobs, * yobs, * zobs;
    int Nx_imgwidth, Ny_imgheight;
    float delta_frequency;
    float left, right, top, bottom;

    float min_eff_idx, total_proj_length;

    /* Subsection B: these are computed from the matlab inputs */
    int Npulses, Nrangebins;
    float c__4_delta_freq;
    float pi_4_f0__clight;

    /* Subsection C: these are CUDA-specific options */
    int device, blockwidth, blockheight;

    /* Subsection D: these are output variables */
    complex_split host_output_image;
    
    /* Section 2. 
     * Parse Matlab's inputs */
    range_profiles.real = (float*)mxGetPr(prhs[0]);
    range_profiles.imag = (float*)mxGetPi(prhs[0]); 

    start_frequencies = (float*)mxGetPr(prhs[1]);
    aimpoint_ranges   = (float*)mxGetPr(prhs[2]);
    xobs              = (float*)mxGetPr(prhs[3]);
    yobs              = (float*)mxGetPr(prhs[4]);
    zobs              = (float*)mxGetPr(prhs[5]);

    Nx_imgwidth     =   (int)mxGetScalar(prhs[6]);
    Ny_imgheight    =   (int)mxGetScalar(prhs[7]);
    delta_frequency = (float)mxGetScalar(prhs[8]);

    left   = (float)mxGetScalar(prhs[ 9]);
    right  = (float)mxGetScalar(prhs[10]);
    bottom = (float)mxGetScalar(prhs[11]);
    top    = (float)mxGetScalar(prhs[12]);

    /* Section 3.
     * Set up some intermediate values */

    /* Range profile dimensions */
    Npulses    = mxGetN(prhs[0]);
    Nrangebins = mxGetM(prhs[0]);
    
    if (nrhs <= 15) {
        min_eff_idx       = (float)mxGetScalar(prhs[13]);
        total_proj_length = (float)mxGetScalar(prhs[14]);
    }
    else {
        min_eff_idx = 0;
        total_proj_length = Nrangebins;
    }


    /* CUDA parameters
     * FIXME: these should only be preset if Matlab didn't specify them */
    device      = 0;
    blockwidth  = BLOCKWIDTH;
    blockheight = BLOCKHEIGHT;
    if (MEXDEBUG) {
        printf("WARNING: CUDA parameters not provided. Auto-selecting:\n"
                "device      %d\n"
                "blockwidth  %d\n"
                "blockheight %d\n", device, blockwidth, blockheight);
    }

    /* Various collection-specific constants */

    c__4_delta_freq = CLIGHT / (4.0f*delta_frequency);

    /* FIXME: this TOTALLY prevents variable start frequency!!!! */
    pi_4_f0__clight = PI*4.0f*extract_f0(start_frequencies, Npulses) / CLIGHT;
    convert_f0(start_frequencies, Npulses);
    


    /* Section 4.
     * Set up Matlab outputs */
    plhs[0] = mxCreateNumericMatrix(Ny_imgheight, Nx_imgwidth, 
            FLOAT_CLASS, mxCOMPLEX);
    host_output_image.real = (float*)mxGetPr(plhs[0]);
    host_output_image.imag = (float*)mxGetPi(plhs[0]);


    /* Section 5.
     * Call Hartley's GPU initialization & invokation code */
    run_bp(range_profiles, xobs, yobs, zobs, 
            aimpoint_ranges, 
            Npulses, Nrangebins, Nx_imgwidth, Ny_imgheight,
            blockwidth, blockheight,
            device,
            host_output_image,
            0, Ny_imgheight,
            c__4_delta_freq, pi_4_f0__clight,
            start_frequencies, left, right, bottom, top, min_eff_idx, total_proj_length);
            
    
    return;
}



void from_gpu_complex_to_bp_complex_split(float2 * data, bp_complex_split out, int size) {
	int i;
	for (i = 0; i < size; i++) {
		out.real[i] = data[i].x;
		out.imag[i] = data[i].y;
	}
}

float2 * format_complex_to_columns(bp_complex_split a, int width_orig, int height_orig) {
	float2 * out = (float2 *) malloc(width_orig * height_orig * sizeof(float2));
	int i, j;
	for (i = 0; i < height_orig; i++) {
		int origOffset = i * width_orig;
		for (j = 0; j < width_orig; j++) {
			int newOffset = j * height_orig;
			out[newOffset + i].x = a.real[origOffset + j];
			out[newOffset + i].y = a.imag[origOffset + j];
		}
	}
	return out;
}

float2 * format_complex(bp_complex_split a, int size) {
	float2 * out = (float2 *) malloc(size * sizeof(float2));
	int i;
	for (i = 0; i < size; i++) {
		out[i].x = a.real[i];
		out[i].y = a.imag[i];
	}
	return out;
}

float4 * format_x_y_z_r(float * x, float * y, float * z, float * r, int size) {
	float4 * out = (float4 *) malloc(size * sizeof(float4));
	int i;
	for (i = 0; i < size; i++) {
		out[i].x = x[i];
		out[i].y = y[i];
		out[i].z = z[i];
		out[i].w = r[i];
	}
	return out;
}



void run_bp(bp_complex_split phd, float * xObs, float * yObs, float * zObs, float * r,
	int my_num_phi, int my_proj_length, int nxout, int nyout, int image_chunk_width,
	int image_chunk_height, int device, bp_complex_split host_output_image,
	int start_output_index, int num_output_rows,
    float c__4_delta_freq, float pi_4_f0__clight, float * start_frequencies,
    float left, float right, float bottom, float top, 
    float min_eff_idx, float total_proj_length) {
	
	MY_CUDA_SAFE_CALL(hipSetDevice(device));

#if ZEROCOPY
    MY_CUDA_SAFE_CALL(hipSetDeviceFlags(hipDeviceMapHost));
#endif

	int num_out_bytes = 2 * sizeof(float) * num_output_rows * nyout;
	float2 * out_image;


	// Set up platform data texture
	float4 * trans_tex_platform_info = format_x_y_z_r(xObs, yObs, zObs, r, my_num_phi);
	hipChannelFormatDesc float4desc = hipCreateChannelDesc<float4>();
	hipArray* array_tex_platform_info;

	MY_CUDA_SAFE_CALL(hipMallocArray( &array_tex_platform_info, &float4desc, 
        my_num_phi, 1));
	MY_CUDA_SAFE_CALL(hipMemcpyToArray(array_tex_platform_info, 0, 0, 
        trans_tex_platform_info, my_num_phi * 4
		* sizeof(float), hipMemcpyHostToDevice));

	tex_platform_info.addressMode[0] = hipAddressModeClamp;
	tex_platform_info.addressMode[1] = hipAddressModeClamp;
	tex_platform_info.filterMode = hipFilterModePoint;
	tex_platform_info.normalized = false; // access with normalized texture coordinates

	MY_CUDA_SAFE_CALL(hipBindTextureToArray(tex_platform_info, array_tex_platform_info, float4desc));

	// Set up input projections texture
	float2 * projections = format_complex_to_columns(phd, my_proj_length, my_num_phi);
	
	hipChannelFormatDesc float2desc = hipCreateChannelDesc<float2>();
	hipArray* cu_proj;

	MY_CUDA_SAFE_CALL(hipMallocArray( &cu_proj, &float2desc, my_num_phi, my_proj_length));
	MY_CUDA_SAFE_CALL(hipMemcpyToArray(cu_proj, 0, 0, projections, my_num_phi
		* my_proj_length * 2 * sizeof(float), hipMemcpyHostToDevice));

	tex_projections.addressMode[0] = hipAddressModeClamp;
	tex_projections.addressMode[1] = hipAddressModeClamp;
	tex_projections.filterMode = hipFilterModeLinear;
	tex_projections.normalized = false; // access with normalized texture coordinates

	MY_CUDA_SAFE_CALL(hipBindTextureToArray(tex_projections, cu_proj, float2desc));

	// Set up and run the kernel
	dim3 dimBlock(image_chunk_width, image_chunk_height, 1);
	dim3 dimGrid(nxout/image_chunk_width, num_output_rows/image_chunk_height);

	float delta_pixel_x = (right-left) / (nxout-1);
	float delta_pixel_y = (top-bottom) / (nyout-1);
	float r_start_pre = (c__4_delta_freq*(float)total_proj_length/((float)total_proj_length-1.0f));
    
    float * device_start_frequencies;
	MY_CUDA_SAFE_CALL(hipMalloc((void**) &device_start_frequencies, sizeof(float)*my_num_phi));
    MY_CUDA_SAFE_CALL(hipMemcpy(device_start_frequencies, start_frequencies, sizeof(float)*my_num_phi, hipMemcpyHostToDevice));


    clock_t c0, c1; 
    c0 = clock();
        
    float * debug_1, * debug_2, *debug_3, *debug_4;

#if ZEROCOPY
	MY_CUDA_SAFE_CALL(hipHostAlloc((void**) &out_image, num_out_bytes, 
        hipHostMallocMapped));

    float2 * device_pointer;
    MY_CUDA_SAFE_CALL(hipHostGetDevicePointer((void **)&device_pointer, 
        (void *)out_image, 0));
    
	backprojection_loop<<<dimGrid, dimBlock>>>(device_pointer, my_num_phi, nyout, delta_pixel_x, delta_pixel_y,
		r_start_pre, total_proj_length, 0, start_output_index,
        c__4_delta_freq, device_start_frequencies, left, bottom, min_eff_idx, trans_tex_platform_info, 
        debug_1,debug_2,debug_3,debug_4,
        0,0);
#else

	MY_CUDA_SAFE_CALL(hipMalloc((void**) &out_image, num_out_bytes));
	backprojection_loop<<<dimGrid, dimBlock>>>(out_image, my_num_phi, nyout, 
        delta_pixel_x, delta_pixel_y,
		r_start_pre, total_proj_length, 0, start_output_index,
        c__4_delta_freq, device_start_frequencies, left, bottom, min_eff_idx, trans_tex_platform_info,
        debug_1, debug_2,debug_3,debug_4,
        0,0);
#endif



    hipError_t this_error = hipGetLastError();
    if ( this_error != hipSuccess) {
        printf("\nERROR: hipGetLastError did NOT return success! DO NOT TRUST RESULTS!\n");
        printf("         '%s'\n", hipGetErrorString(this_error) );
    }
 
    if ( hipDeviceSynchronize() != hipSuccess)
        printf("\nERROR: threads did NOT synchronize! DO NOT TRUST RESULTS!\n\n");
    c1 = clock();
    printf("INFO: CUDA-mex kernel took %f s\n", (float) (c1 - c0)/CLOCKS_PER_SEC);

#if ZEROCOPY
	from_gpu_complex_to_bp_complex_split(out_image, host_output_image, num_output_rows * nyout);
    MY_CUDA_SAFE_CALL(hipHostFree(out_image));
#else
	float2 * host_data = (float2 *) malloc(num_out_bytes);
	//double start_t = -ms_walltime();
	MY_CUDA_SAFE_CALL(hipMemcpy(host_data, out_image, num_out_bytes, hipMemcpyDeviceToHost));
	//printf("MEMCPY,%lf\n", (start_t + ms_walltime()));
	from_gpu_complex_to_bp_complex_split(host_data, host_output_image, num_output_rows
		* nyout);
	free(host_data);
	hipFree(out_image);
#endif
    hipFree(device_start_frequencies);
	free(trans_tex_platform_info);
	free(projections);

	hipFreeArray(array_tex_platform_info);
	hipFreeArray(cu_proj);

    MY_CUDA_SAFE_CALL(hipDeviceReset());

}

__global__ void testing_platform_tex(float * x, float * y, float * z, float * w, float num)
{
    float4 foo = tex1D(tex_platform_info, num);
    x[0] = foo.x;
    y[0] = foo.y;
    z[0] = foo.z;
    w[0] = foo.w;
}

__global__ void testing_platform(float4 * plat, float * xx, float * yy, float * zz, float * ww, int num)
{
    float4 foo = plat[num];
    xx[0] = foo.x;
    yy[0] = foo.y;
    zz[0] = foo.z;
    ww[0] = foo.w;
}

__global__ void testing_proj_tex(float * re, float * im, float xx, float yy)
{
    float2 foo = tex2D(tex_projections, xx, yy); // x: proj num, y: rbin
    re[0] = foo.x;
    im[0] = foo.y;
}

__global__ void testing_r(float xpixel, float ypixel, float xa, float ya, float za, float * R) 
{
    (*R) = ( CAREFUL_AMINUSB_SQ(xpixel, xa) + CAREFUL_AMINUSB_SQ(ypixel, ya) + 
            __fmul_rn(za, za));
}



/* Main kernel.
 *
 * Tuning options:
 * - is it worth #defining radar parameters like start_frequency?
 *      ............  or imaging parameters like xmin/ymax?
 * - Make sure (4 pi / c) is computed at compile time!
 * - Use 24-bit integer multiplications!
 *
 * */
__global__ void backprojection_loop(float2 * full_image,
        int Nphi, int nyout, float delta_pixel_x, float delta_pixel_y, 
        float R_START_PRE, int PROJ_LENGTH,
        int X_OFFSET, int Y_OFFSET,
        float C__4_DELTA_FREQ, float  * PI_4_F0__CLIGHT, 
        float left, float bottom, float min_eff_idx, float4 * platform_info,
        float * debug_effective_idx, float * debug_2, float * x_mat, float * y_mat, 
        float rmin, float rmax) {

    float2 subimage;
    subimage = make_float2(0.0f, 0.0f);
    float2 csum; // For compensated sum
    float y, t;
    csum = make_float2(0.0f, 0.0f);

    float xpixel = left   + (float)(blockIdx.x * BLOCKWIDTH  + threadIdx.x) * 
        delta_pixel_x;
    float ypixel = bottom + (float)(blockIdx.y * BLOCKHEIGHT + threadIdx.y) * 
        delta_pixel_y;
    
    float2 texel;

    __shared__ int proj_num;
    __shared__ float4 platform;
    __shared__ int copyblock;

    __shared__ float delta_r;
    delta_r = rmax - rmin;
    __shared__ float Nl1_dr;
    Nl1_dr = __fdiv_rn((float)PROJ_LENGTH - 1.0f, delta_r);

    copyblock = (blockIdx.y * BLOCKHEIGHT) * nyout + blockIdx.x * BLOCKWIDTH;

    /* Now, let's loop through these projections! 
     * */
#pragma unroll 3
    for (proj_num=0; proj_num < Nphi; ++proj_num) {

        //platform = tex1D(tex_platform_info, (float)proj_num + 0.5f);
        platform = platform_info[proj_num];

        /* R_reciprocal = 1/R = 1/sqrt(sum_{# in xyz} [#pixel - #platform]^2),
         * This is the distance between the platform and every pixel.
         */
         /*
        float R = sqrtf( 
                (xpixel - platform.x) * 
                (xpixel - platform.x) +
                (ypixel - platform.y) * 
                (ypixel - platform.y) +
                platform.z * platform.z);*/
        float R = MAKERADIUS(xpixel, ypixel, platform.x, platform.y, platform.z);

        /* Per-pixel-projection phasor = exp(1j 4 pi/c * f_min * R). */
        //float2 pixel_scale = expjf_div_2(PI_4_F0__CLIGHT[proj_num] * R * 0.5f);
        float2 pixel_scale = expjf(PI_4_F0__CLIGHT[proj_num] * R);
        
        /* The fractional range bin for this pixel, this projection */
        /*
        float effective_idx = ((float)PROJ_LENGTH-1.0f) *
            (R - ( platform.w - R_START_PRE )) / (2.0f*C__4_DELTA_FREQ) 
            - min_eff_idx;*/
        //float effective_idx = ((float)PROJ_LENGTH-1.0f) / (rmax - rmin) * (R - platform.w - rmin);
        float effective_idx = __fmul_rn(Nl1_dr , __fadd_rn(__fadd_rn(R, -1.0f*platform.w), -1.0f*rmin));

        /* This is the interpolated range profile element for this pulse */

        // Flipped textres
        /*texel = tex2D(tex_projections, 
                0.5f+effective_idx, 0.5f+(float)proj_num);*/
        // offset textures
        texel = tex2D(tex_projections, 0.5f+(float)proj_num, 0.5f+effective_idx);

        /* Scale "texel" by "pixel_scale".
           The RHS of these 2 lines just implement complex multiplication.
        */
        y = REAL(texel)*REAL(pixel_scale) - REAL(csum);
        t = subimage.x + y;
        csum.x = (t-subimage.x) - y;
        subimage.x = t;

        y = -1.0f*IMAG(texel)*IMAG(pixel_scale) - REAL(csum);
        t = subimage.x + y;
        csum.x = (t-subimage.x) - y;
        subimage.x = t;

        y = REAL(texel)*IMAG(pixel_scale) - IMAG(csum);
        t = subimage.y + y;
        csum.y = (t-subimage.y) - y;
        subimage.y = t;

        y = IMAG(texel)*REAL(pixel_scale) - IMAG(csum);
        t = subimage.y + y;
        csum.y = (t-subimage.y) - y;
        subimage.y = t;

        /*
        subimage.x += REAL(texel)*REAL(pixel_scale) - 
                IMAG(texel)*IMAG(pixel_scale);
        subimage.y += REAL(texel)*IMAG(pixel_scale) + 
                IMAG(texel)*REAL(pixel_scale);
        */

        if (proj_num==0) {
            debug_effective_idx[copyblock + (threadIdx.y) * nyout + threadIdx.x] = effective_idx;
            debug_2[copyblock + (threadIdx.y) * nyout + threadIdx.x] = R;
            x_mat[copyblock + (threadIdx.y) * nyout + threadIdx.x] = platform.x;
            y_mat[copyblock + (threadIdx.y) * nyout + threadIdx.x] = platform.y;
        }
    }
    /* Copy this thread's pixel back to global memory */
    //full_image[(blockIdx.y * BLOCKHEIGHT + threadIdx.y) * nyout + 
    //    blockIdx.x * BLOCKWIDTH + threadIdx.x] = subimage;
    full_image[copyblock + (threadIdx.y) * nyout + threadIdx.x] = subimage;
}


/* Credits: from BackProjectionKernal.c: "originally by reinke".
 * Given a float X, returns float2 Y = exp(j * X).
 *
 * __device__ code is always inlined. */
__device__ 
float2 expjf(float in) {
    float2 out;
    float t, tb;
#if USE_FAST_MATH
    t = __tanf(in / 2.0f);
#else
    t = tan(in / 2.0f);
#endif
    tb = t*t + 1.0f;
    out.x = (2.0f - tb) / tb; /* Real */
    out.y = (2.0f * t) / tb; /* Imag */
    return out;
}

__device__ 
float2 expjf_div_2(float in) {
    float2 out;
    float t, tb;
    //t = __tanf(in - (float)((int)(in/(PI2)))*PI2 );
    t = __tanf(in - PI * rintf(in/PI) );
    tb = t*t + 1.0f;
    out.x = (2.0f - tb) / tb; /* Real */
    out.y = (2.0f * t) / tb; /* Imag */
    return out;
}


