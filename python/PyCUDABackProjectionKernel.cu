#include "hip/hip_runtime.h"
/* vim: set tabstop=4, shiftwidth=4, expandtab */
#include "hip/hip_runtime.h"   /* CUDA */

/***
 * Compiler logics
 * **/
#define CLIGHT 299792458.0f        /* c: speed of light, m/s */
#define PI 3.141592653589793116f   /* pi, accurate to 128-bits */
#define PI2 6.283185307179586232f  /* 2*pi */
#define PI_4__CLIGHT (4.0f * PI / CLIGHT)

#define REAL(vec) (vec.x)
#define IMAG(vec) (vec.y)


/* The following macros are defined for common arithmetic operations. We use
 * the __f*_rn() CUDA intrinsics in order to prevent these operations from
 * being performed on FMADs, whose implementation forces the truncation of the
 * result between the multiple and add, giving us less accurate results.
*/
#define CAREFUL_AMINUSB_SQ(x,y) __fmul_rn(__fadd_rn((x), -1.0f*(y)), __fadd_rn((x), -1.0f*(y)))
#define MAKERADIUS(xpixel,ypixel, xa,ya,za) sqrtf(CAREFUL_AMINUSB_SQ(xpixel, xa) + CAREFUL_AMINUSB_SQ(ypixel, ya) + __fmul_rn(za, za))

#define BLOCKWIDTH    16
#define BLOCKHEIGHT   16


/***
 * Type defs
 * ***/
typedef float FloatType; /* FIXME: this should be used everywhere */


/***
 * Prototypes
 * ***/

__device__ float2 expjf(float in);
__device__ float2 expjf_div_2(float in);


/* Complex textures containing range profiles */
texture<float2, 2, hipReadModeElementType> tex_projections;   


/* Main kernel.
 * */
__global__ void backprojection_loop(float2 * full_image,
        int Nphi, int IMG_HEIGHT, float delta_pixel_x, float delta_pixel_y, 
        int PROJ_LENGTH,
        float  * PI_4_F0__CLIGHT, 
        float LEFT, float BOTTOM, 
        float4 * PLATFORM_INFO,
        float rmin, float rmax) {

    float2 subimage;
    subimage = make_float2(0.0f, 0.0f);
    float2 csum; // For compensated sum
    float y, t;
    csum = make_float2(0.0f, 0.0f);

    float xpixel = LEFT + (float)(blockIdx.x * BLOCKWIDTH  + threadIdx.x) * 
        delta_pixel_x;
    float ypixel = BOTTOM + (float)(blockIdx.y * BLOCKHEIGHT + threadIdx.y) * 
        delta_pixel_y;
    
    float2 texel;

    __shared__ int proj_num;
    __shared__ float4 platform;
    __shared__ int copyblock;

    __shared__ float delta_r;
    delta_r = rmax - rmin;
    __shared__ float Nl1_dr;
    Nl1_dr = __fdiv_rn((float)PROJ_LENGTH - 1.0f, delta_r);

    copyblock = (blockIdx.y * BLOCKHEIGHT) * IMG_HEIGHT + blockIdx.x * BLOCKWIDTH;

    /* Now, let's loop through these projections! 
     * */
#pragma unroll 3
    for (proj_num=0; proj_num < Nphi; ++proj_num) {
        platform = PLATFORM_INFO[proj_num];

        /* R_reciprocal = 1/R = 1/sqrt(sum_{# in xyz} [#pixel - #platform]^2),
         * This is the distance between the platform and every pixel.
         */
         /* // This is what's *really* happening.
        float R = sqrtf( 
                (xpixel - platform.x) * 
                (xpixel - platform.x) +
                (ypixel - platform.y) * 
                (ypixel - platform.y) +
                platform.z * platform.z);*/
        float R = MAKERADIUS(xpixel, ypixel, platform.x, platform.y, platform.z);

        /* Per-pixel-projection phasor = exp(1j 4 pi/c * f_min * R). */
        // Faster, less accurate:
        // float2 pixel_scale = expjf_div_2(PI_4_F0__CLIGHT[proj_num] * R * 0.5f);
        // Slower, more accurate: 
        float2 pixel_scale = expjf(PI_4_F0__CLIGHT[proj_num] * R);
        
        /* The fractional range bin for this pixel, this projection. Implements
         * effective_idx = Nl1_dr*( R-platform.w-rmin ) 
         */
        float effective_idx = __fmul_rn(Nl1_dr , __fadd_rn(__fadd_rn(R, -1.0f*platform.w), -1.0f*rmin));

        /* This is the interpolated range profile element for this pulse */
        texel = tex2D(tex_projections, 0.5f+(float)proj_num, 0.5f+effective_idx);

        /* Scale "texel" by "pixel_scale".
           The RHS of these 2 lines just implement complex multiplication.
        */
        /*
        subimage.x += REAL(texel)*REAL(pixel_scale) - 
                IMAG(texel)*IMAG(pixel_scale);
        subimage.y += REAL(texel)*IMAG(pixel_scale) + 
                IMAG(texel)*REAL(pixel_scale);
        */

        // I'm experimenting with cumulative summing to try and reduce errors.
        // See http://en.wikipedia.org/wiki/Kahan_summation_algorithm
        y = REAL(texel)*REAL(pixel_scale) - REAL(csum);
        t = subimage.x + y;
        csum.x = (t-subimage.x) - y;
        subimage.x = t;

        y = -1.0f*IMAG(texel)*IMAG(pixel_scale) - REAL(csum);
        t = subimage.x + y;
        csum.x = (t-subimage.x) - y;
        subimage.x = t;

        y = REAL(texel)*IMAG(pixel_scale) - IMAG(csum);
        t = subimage.y + y;
        csum.y = (t-subimage.y) - y;
        subimage.y = t;

        y = IMAG(texel)*REAL(pixel_scale) - IMAG(csum);
        t = subimage.y + y;
        csum.y = (t-subimage.y) - y;
        subimage.y = t;

    }
    /* Copy this thread's pixel back to global memory */
    //full_image[(blockIdx.y * BLOCKHEIGHT + threadIdx.y) * IMG_HEIGHT + 
    //    blockIdx.x * BLOCKWIDTH + threadIdx.x] = subimage;
    full_image[copyblock + (threadIdx.y) * IMG_HEIGHT + threadIdx.x] = subimage;
}


/* Credits: from BackProjectionKernal.c: "originally by reinke".
 * Given a float X, returns float2 Y = exp(j * X).
 *
 * __device__ code is always inlined. */
__device__ 
float2 expjf(float in) {
    float2 out;
    float t, tb;
    t = tan(in / 2.0f);
    tb = t*t + 1.0f;
    out.x = (2.0f - tb) / tb; /* Real */
    out.y = (2.0f * t) / tb; /* Imag */
    return out;
}


/* This is derived from the above function, but is way faster because it uses
 * hardware __tanf(). This is allowable only if the argument to it is
 * "small"---within +/- 2\pi. 
 *
 * Given float X, returns exp(j * 2*X).
*/
__device__ 
float2 expjf_div_2(float in) {
    float2 out;
    float t, tb;
    //t = __tanf(in - (float)((int)(in/(PI2)))*PI2 );
    t = __tanf(in - PI * rintf(in/PI) );
    tb = t*t + 1.0f;
    out.x = (2.0f - tb) / tb; /* Real */
    out.y = (2.0f * t) / tb; /* Imag */
    return out;
}


