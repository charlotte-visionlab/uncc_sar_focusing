#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2022 Andrew R. Willis
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

// Standard Library includes
#include <iomanip>
#include <sstream>
#include <fstream>

#include <third_party/log.h>
#include <third_party/cxxopts.hpp>

#include <cuGridSearch.cuh>

// this declaration needs to be in any C++ compiled target for CPU
//#define CUDAFUNCTION

#include <charlotte_sar_api.hpp>
#include <uncc_sar_globals.hpp>

#include <uncc_sar_focusing.hpp>
#include <uncc_sar_matio.hpp>

#include "../gpuBackProjection/cuda_sar_focusing/cuda_sar_focusing.hpp"

#include "gridSearchErrorFunctions.cuh"

typedef float NumericType;

#define grid_dimension_quadratic 9        // the dimension of the grid, e.g., 1 => 1D grid, 2 => 2D grid, 3=> 3D grid, etc.
#define grid_dimension_linear 6        // the dimension of the grid, e.g., 1 => 1D grid, 2 => 2D grid, 3=> 3D grid, etc.

typedef float grid_precision;   // the type of values in the grid, e.g., float, double, int, etc.
typedef float func_precision;   // the type of values taken by the error function, e.g., float, double, int, etc.
typedef float pixel_precision; // the type of values in the image, e.g., float, double, int, etc.

// TODO: THIS WILL NEED TO BE CHANGED TO FIT THE ERROR FUNCTION (Look at changes to error function)

typedef func_byvalue_t<func_precision, grid_precision, grid_dimension_linear,
        hipfftComplex*,
        int, int,
        NumericType, NumericType,
        NumericType, NumericType,
        NumericType, NumericType,
        NumericType*,
        NumericType*,
        NumericType*,
        NumericType*,
        NumericType*,
        SAR_ImageFormationParameters<NumericType>*,
        NumericType* > image_err_func_byvalue_linear;

typedef func_byvalue_t<func_precision, grid_precision, grid_dimension_quadratic,
        hipfftComplex*,
        int, int,
        NumericType, NumericType,
        NumericType, NumericType,
        NumericType, NumericType,
        NumericType*,
        NumericType*,
        NumericType*,
        NumericType*,
        NumericType*,
        SAR_ImageFormationParameters<NumericType>*,
        NumericType* > image_err_func_byvalue_quadratic;

// TODO: THIS WILL ALSO NEED TO BE CHANGED TO FIT THE ERROR FUNCTION
__device__ image_err_func_byvalue_linear dev_func_byvalue_ptr_linear = kernelWrapper<func_precision, grid_precision, grid_dimension_linear, NumericType>;
__device__ image_err_func_byvalue_quadratic dev_func_byvalue_ptr_quadratic = kernelWrapper<func_precision, grid_precision, grid_dimension_quadratic, NumericType>;


template<typename __nTp>
std::vector<__nTp> vectorDiff(std::vector<__nTp> values) {
    std::vector<__nTp> temp;
    for (int i = 0; i < values.size()-1; i++)
        temp.push_back(values[i+1] - values[i]);

    return temp;
}

template<typename __nTp>
std::vector<__nTp> generateDiffEstimate(__nTp slope, __nTp constant, int N) {
    std::vector<__nTp> temp;
    for(int i = 0; i < N; i++)
        temp.push_back(slope*i+constant);
    return temp;
}

template<typename __nTp>
std::vector<__nTp> vectorAppendCumSum(__nTp start, std::vector<__nTp> values) {
    std::vector<__nTp> temp;
    __nTp sum = start;
    temp.push_back(start);
    for (int i = 0; i < values.size(); i++){
        sum += values[i];
        temp.push_back(sum);
    }

    return temp;
}

template<typename __nTp>
void bestFit(__nTp* coeffs, std::vector<__nTp> values, int nPulse) {
    // double sumX = 0.0;
    // double sumY = 0.0;
    // double N = values.size();
    // double sumXY = 0.0;
    // double sumXX = 0.0;

    // for (int i =0; i < values.size(); i++) {
    //     sumX += (__nTp)i;
    //     sumY += values[i];
    //     sumXY += ((__nTp)i*values[i]);
    //     sumXX += ((__nTp)i * (__nTp)i);
    // }

    // double numS = N * sumXY - sumX * sumY;
    // double den = N * sumXX - sumX * sumX;

    // double numC = sumY * sumXX - sumX * sumXY;
    
    // double temp1 = numS/den;
    // double temp2 = numC/den;

    // float tempa = (float)temp1;
    // float tempb = (float)temp2;

    // coeffs[0] = 0;
    // coeffs[1] = tempa;
    // coeffs[2] = tempb;

    int numN = nPulse;
    double N = numN;
    double x1 = 0;
    double x2 = 0;
    double f0 = 0;
    double f1 = 0;

    for (int i = 0; i < numN; i++) {
        x1 += i;
        x2 += i * i;
        f0 += values[i];
        f1 += values[i] * i;
    }

    double D = -1 * x1 * x1 + N * x2;

    double a = N * f1 - f0 * x1;
    double b = f0 * x2 - f1 * x1;
    printf("N = %f\nx1 = %f\nx2 = %f\nf0 = %f\nf1 = %f\n", N, x1, x2, f0, f1);
    printf("a = %f\nb = %f\n D = %f\n", a, b, D);
    a /= D;
    b /= D;
    printf("a1 = %f\nb2 = %f\n", a, b);
    float temp_a = (float)a;
    float temp_b = (float)b;
    printf("temp_a = %f\ntemp_b = %f\n", temp_a, temp_b);
    coeffs[0] = temp_b;
    coeffs[1] = temp_a;
}

template<typename __nTp>
void quadFit(__nTp* coeffs, std::vector<__nTp> values, int nPulse) {
    int numN = nPulse;
    double N = numN;
    double x1 = 0;
    double x2 = 0;
    double x3 = 0;
    double x4 = 0;
    double f0 = 0;
    double f1 = 0;
    double f2 = 0;

    for (int i = 0; i < numN; i++) {
        x1 += i;
        x2 += i * i;
        x3 += i * i * i;
        x4 += i * i * i * i;
        f0 += values[i];
        f1 += values[i] * i;
        f2 += values[i] * i * i;
    }

    double D = x4 * x1 * x1 - 2 * x1 * x2 * x3 + x2 * x2 * x2 - N * x4 * x2 + N * x3 * x3;

    double a = f2 * x1 * x1 - f1 * x1 * x2 - f0 * x3 * x1 + f0 * x2 * x2 - N * f2 * x2 + N * f1 * x3;
    double b = f1 * x2 * x2 - N * f1 *x4 + N * f2 * x3 + f0 * x1 * x4 - f0 * x2 * x3 - f2 * x1 * x2;
    double c = f2 * x2 * x2 - f1 * x2 * x3 - f0 * x4 * x2 + f0 * x3 * x3 - f2 * x1 * x3 + f1 * x1 * x4;

    a /= D;
    b /= D;
    c /= D;

    float temp_a = (float)a;
    float temp_b = (float)b;
    float temp_c = (float)c;
    
    coeffs[0] = temp_c;
    coeffs[1] = temp_b;
    coeffs[2] = temp_a;
}

// TODO: Need to work on setting up the grid search

template <typename __nTp, typename __nTpParams>
void grid_cuda_focus_SAR_image(const SAR_Aperture<__nTp>& sar_data,
        const SAR_ImageFormationParameters<__nTpParams>& sar_image_params,
        CArray<__nTp>& output_image, std::ofstream* myfile, int multiRes, int style) {

    switch (sar_image_params.algorithm) {
        case SAR_ImageFormationParameters<__nTpParams>::ALGORITHM::BACKPROJECTION:
            std::cout << "Selected backprojection algorithm for focusing." << std::endl;
            //run_bp(sar_data, sar_image_params, output_image);
            break;
        case SAR_ImageFormationParameters<__nTpParams>::ALGORITHM::MATCHED_FILTER:
            std::cout << "Selected matched filtering algorithm for focusing." << std::endl;
            //run_mf(SARData, SARImgParams, output_image);
            //break;
        default:
            std::cout << "focus_SAR_image()::Algorithm requested is not recognized or available." << std::endl;
            return;
    }

    // Display maximum scene size and resolution
    std::cout << "Maximum Scene Size:  " << std::fixed << std::setprecision(2) << sar_image_params.max_Wy_m << " m range, "
            << sar_image_params.max_Wx_m << " m cross-range" << std::endl;
    std::cout << "Maximum Resolution:  " << std::fixed << std::setprecision(2) << sar_image_params.slant_rangeResolution << "m range, "
            << sar_image_params.azimuthResolution << " m cross-range" << std::endl;
    GPUMemoryManager cuda_res;

    if (initialize_GPUMATLAB(cuda_res.deviceId) == EXIT_FAILURE) {
        std::cout << "cuda_focus_SAR_image::Could not initialize the GPU. Exiting..." << std::endl;
        return;
    }

    if (initialize_CUDAResources(sar_data, sar_image_params, cuda_res) == EXIT_FAILURE) {
        std::cout << "cuda_focus_SAR_image::Problem found initializing resources on the GPU. Exiting..." << std::endl;
        return;
    }

    // Calculate range bins for range compression-based algorithms, e.g., backprojection
    RangeBinData<__nTp> range_bin_data;
    range_bin_data.rangeBins.shape.push_back(sar_image_params.N_fft);
    range_bin_data.rangeBins.shape.push_back(1);
    range_bin_data.rangeBins.data.resize(sar_image_params.N_fft);
    __nTp* rangeBins = &range_bin_data.rangeBins.data[0]; //[sar_image_params.N_fft];
    __nTp minRange = range_bin_data.minRange;
    __nTp maxRange = range_bin_data.maxRange;

    minRange = std::numeric_limits<float>::infinity();
    maxRange = -std::numeric_limits<float>::infinity();
    for (int rIdx = 0; rIdx < sar_image_params.N_fft; rIdx++) {
        // -maxWr/2:maxWr/Nfft:maxWr/2
        //float rVal = ((float) rIdx / Nfft - 0.5f) * maxWr;
        __nTp rVal = RANGE_INDEX_TO_RANGE_VALUE(rIdx, sar_image_params.max_Wy_m, sar_image_params.N_fft);
        rangeBins[rIdx] = rVal;
        if (minRange > rangeBins[rIdx]) {
            minRange = rangeBins[rIdx];
        }
        if (maxRange < rangeBins[rIdx]) {
            maxRange = rangeBins[rIdx];
        }
    }

    cuda_res.copyToDevice("range_vec", (void *) &range_bin_data.rangeBins.data[0],
            range_bin_data.rangeBins.data.size() * sizeof (range_bin_data.rangeBins.data[0]));

    std::cout << cuda_res << std::endl;
    int numSamples = sar_data.sampleData.data.size();
    int newSize = pow(2, ceil(log(sar_data.sampleData.data.size()) / log(2)));

    clock_t c0, c1, c2;

    c0 = clock();
    //std::cout << printf("N_fft: %d, numAzimuthSamples: %d, numSamples: %d\n\n",sar_image_params.N_fft, sar_data.numAzimuthSamples, newSize);
    cuifft(cuda_res.getDeviceMemPointer<hipfftComplex>("sampleData"), sar_image_params.N_fft, sar_data.numAzimuthSamples);
    cufftNormalize_1DBatch(cuda_res.getDeviceMemPointer<hipfftComplex>("sampleData"), sar_image_params.N_fft, sar_data.numAzimuthSamples);
    cufftShift_1DBatch<hipfftComplex>(cuda_res.getDeviceMemPointer<hipfftComplex>("sampleData"), sar_image_params.N_fft, sar_data.numAzimuthSamples);
    c1 = clock();
    printf("INFO: CUDA FFT kernels took %f ms.\n", (float) (c1 - c0) * 1000 / CLOCKS_PER_SEC);

    __nTp delta_x_m_per_pix = sar_image_params.Wx_m / (sar_image_params.N_x_pix - 1);
    __nTp delta_y_m_per_pix = sar_image_params.Wy_m / (sar_image_params.N_y_pix - 1);
    __nTp left_m = sar_image_params.x0_m - sar_image_params.Wx_m / 2;
    __nTp bottom_m = sar_image_params.y0_m - sar_image_params.Wy_m / 2;

    // Set up and run the kernel
    dim3 dimBlock(cuda_res.blockwidth, cuda_res.blockheight, 1);
    dim3 dimGrid(std::ceil((float) sar_image_params.N_x_pix / cuda_res.blockwidth),
            std::ceil((float) sar_image_params.N_y_pix / cuda_res.blockheight));
    c0 = clock();

    // LINE FITTING BASED ON PULSE
    float *xCoeffs, *yCoeffs, *zCoeffs;

    // TODO: CHANGE PULSE NUMBERS, 10, 20, 30 for linear

    std::vector<NumericType> xPossDiff = vectorDiff(sar_data.Ant_x.data);
    std::vector<NumericType> yPossDiff = vectorDiff(sar_data.Ant_y.data);
    std::vector<NumericType> zPossDiff = vectorDiff(sar_data.Ant_z.data);

    // Put all of this outside the for loop
    int numRSamples = sar_data.numRangeSamples, numASamples = sar_data.numAzimuthSamples;
    hipfftComplex* data_p = cuda_res.getDeviceMemPointer<hipfftComplex>("sampleData");
    __nTp* ax_p = cuda_res.getDeviceMemPointer<__nTp>("Ant_x");
    __nTp* ay_p = cuda_res.getDeviceMemPointer<__nTp>("Ant_y");
    __nTp* az_p = cuda_res.getDeviceMemPointer<__nTp>("Ant_z");
    __nTp* sr_p = cuda_res.getDeviceMemPointer<__nTp>("slant_range");
    __nTp* sf_p = cuda_res.getDeviceMemPointer<__nTp>("startF");
    SAR_ImageFormationParameters<__nTpParams>* sip_p = cuda_res.getDeviceMemPointer<SAR_ImageFormationParameters < __nTpParams >> ("sar_image_params");
    __nTp* rv_p = cuda_res.getDeviceMemPointer<__nTp>("range_vec");
    hipfftComplex* oi_p = cuda_res.getDeviceMemPointer<hipfftComplex>("output_image");
    checkCudaErrors(hipDeviceSetLimit(hipLimitMallocHeapSize, 1 << 30));

    // GET GRID SEARCH RANGE
    // grid_precision gridDiff = 1e-4f;
    grid_precision gridDiff = 1.3f;
    grid_precision gridN = 11;

    float totalTime = 0;

    if(style == 0) {
        printf("Using Linear Model\n");
        xCoeffs = new float[2];
        yCoeffs = new float[2];
        zCoeffs = new float[2];
        grid_precision minParams[grid_dimension_linear] = {0};

        bestFit<NumericType>(xCoeffs, sar_data.Ant_x.data, sar_data.numAzimuthSamples);
        bestFit<NumericType>(yCoeffs, sar_data.Ant_y.data, sar_data.numAzimuthSamples);
        bestFit<NumericType>(zCoeffs, sar_data.Ant_z.data, sar_data.numAzimuthSamples);

        printf("X - Slope coeff = %f\n    Const coeff = %f\n",xCoeffs[1],xCoeffs[0]);
        printf("Y - Slope coeff = %f\n    Const coeff = %f\n",yCoeffs[1],yCoeffs[0]);
        printf("Z - Slope coeff = %f\n    Const coeff = %f\n",zCoeffs[1],zCoeffs[0]);

        *myfile << "gt," << xCoeffs[1] << ',' << xCoeffs[0] << ','
                << yCoeffs[1] << ',' << yCoeffs[0] << ','
                << zCoeffs[1] << ',' << zCoeffs[0] << ',';

        std::vector<grid_precision> start_point = {(grid_precision) xCoeffs[0], (grid_precision) xCoeffs[1]-gridDiff,
                                                   (grid_precision) yCoeffs[0], (grid_precision) yCoeffs[1]-gridDiff,
                                                   (grid_precision) zCoeffs[0],(grid_precision) zCoeffs[1]-gridDiff};
        std::vector<grid_precision> end_point = {(grid_precision) xCoeffs[0], (grid_precision) xCoeffs[1]+gridDiff,
                                                 (grid_precision) yCoeffs[0], (grid_precision) yCoeffs[1]+gridDiff,
                                                 (grid_precision) zCoeffs[0], (grid_precision) zCoeffs[1]+gridDiff};
        std::vector<grid_precision> grid_numSamples = {(grid_precision)gridN, (grid_precision) gridN,
                                                       (grid_precision)gridN, (grid_precision) gridN,
                                                       (grid_precision)gridN, (grid_precision) gridN};
        image_err_func_byvalue_linear host_func_byval_ptr;
        // Copy device function pointer for the function having by-value parameters to host side
        hipMemcpyFromSymbol(&host_func_byval_ptr, HIP_SYMBOL(dev_func_byvalue_ptr_linear),
                             sizeof(dev_func_byvalue_ptr_linear));

        for(int iii = 0; iii < multiRes; iii++) {
            CudaGrid<grid_precision, grid_dimension_linear> grid;
            ck(hipMalloc(&grid.data(), grid.bytesSize()));

            grid.setStartPoint(start_point);
            grid.setEndPoint(end_point);
            grid.setNumSamples(grid_numSamples);
            grid.display("grid");

            grid_precision axis_sample_counts[grid_dimension_linear];
            grid.getAxisSampleCounts(axis_sample_counts);

            CudaTensor<func_precision, grid_dimension_linear> func_values(axis_sample_counts);
            ck(hipMalloc(&func_values._data, func_values.bytesSize()));

            // first template argument is the error function return type
            // second template argument is the grid point value type
            CudaGridSearcher<func_precision, grid_precision, grid_dimension_linear> gridsearcher(grid, func_values);

            c1 = clock();
            gridsearcher.search_by_value_stream(host_func_byval_ptr, 1000, 451,
                    // gridsearcher.search_by_value(host_func_byval_ptr,
                                                data_p,
                                                numRSamples, numASamples,
                                                delta_x_m_per_pix, delta_y_m_per_pix,
                                                left_m, bottom_m,
                                                minRange, maxRange,
                                                ax_p,
                                                ay_p,
                                                az_p,
                                                sr_p,
                                                sf_p,
                                                sip_p,
                                                rv_p);
            c2 = clock();
            float searchTime = (float) (c2 - c1) * 1000 / CLOCKS_PER_SEC;
            printf("INFO: cuGridSearch took %f ms.\n", searchTime);

            totalTime += searchTime;

            func_precision min_value;
            int32_t min_value_index1d;
            func_values.find_extrema(min_value, min_value_index1d);

            grid_precision min_grid_point[grid_dimension_linear];
            grid.getGridPoint(min_grid_point, min_value_index1d);
            std::cout << "Minimum found at point p = { ";
            for (int d=0; d < grid_dimension_linear; d++) {
                minParams[d] = min_grid_point[d];
                std::cout << min_grid_point[d] << ((d < grid_dimension_linear - 1) ? ", " : " ");

                start_point[d] = min_grid_point[d] - (end_point[d] - start_point[d]) / 4;
                end_point[d] = min_grid_point[d] + (end_point[d] - start_point[d]) / 4;
                // Update grid here or outside (basically somewhere near here)
            }
            std::cout << "}" << std::endl;

            ck(hipFree(grid.data()));
            ck(hipFree(func_values.data()));

        }

        printf("Total time took: %f\n", totalTime);
        *myfile << "time," << totalTime << ',';

        printf("MinParams[");
        for(int d = 0; d < grid_dimension_linear; d++) {
            printf("%e,", minParams[d]);
        }
        printf("]\n");

        *myfile << "found,";
        for(int i = 0; i < grid_dimension_linear; i++)
            *myfile << minParams[i] << ',';

        nv_ext::Vec<grid_precision, grid_dimension_linear> minParamsVec(minParams);
        computeImageKernel<func_precision, grid_precision, grid_dimension_linear, __nTp><<<1,451>>>(minParamsVec,
                                                                                                       data_p,
                                                                                                       numRSamples, numASamples,
                                                                                                       delta_x_m_per_pix, delta_y_m_per_pix,
                                                                                                       left_m, bottom_m,
                                                                                                       minRange, maxRange,
                                                                                                       ax_p,
                                                                                                       ay_p,
                                                                                                       az_p,
                                                                                                       sr_p,
                                                                                                       sf_p,
                                                                                                       sip_p,
                                                                                                       rv_p,
                                                                                                       oi_p);

    } else {
        printf("Using Quadratic Model\n");
        xCoeffs = new float[3];
        yCoeffs = new float[3];
        zCoeffs = new float[3];
        grid_precision minParams[grid_dimension_quadratic] = {0};

        quadFit<NumericType>(xCoeffs, sar_data.Ant_x.data, sar_data.numAzimuthSamples);
        quadFit<NumericType>(yCoeffs, sar_data.Ant_y.data, sar_data.numAzimuthSamples);
        quadFit<NumericType>(zCoeffs, sar_data.Ant_z.data, sar_data.numAzimuthSamples);

        printf("X - Quad coeff = %f\n    Slope coeff = %f\n    Const coeff = %f\n",xCoeffs[2],xCoeffs[1],xCoeffs[0]);
        printf("Y - Quad coeff = %f\n    Slope coeff = %f\n    Const coeff = %f\n",yCoeffs[2],yCoeffs[1],yCoeffs[0]);
        printf("Z - Quad coeff = %f\n    Slope coeff = %f\n    Const coeff = %f\n",zCoeffs[2],zCoeffs[1],zCoeffs[0]);

        *myfile << "gt," << xCoeffs[2] << ',' << xCoeffs[1] << ',' << xCoeffs[0] << ','
                << yCoeffs[2] << ',' << yCoeffs[1] << ',' << yCoeffs[0] << ','
                << zCoeffs[2] << ',' << zCoeffs[1] << ',' << zCoeffs[0] << ',';
        std::vector<grid_precision> start_point = {(grid_precision) xCoeffs[0], (grid_precision) xCoeffs[1]-gridDiff, (grid_precision) xCoeffs[2],
                                                   (grid_precision) yCoeffs[0], (grid_precision) yCoeffs[1]-gridDiff, (grid_precision) yCoeffs[2],
                                                   (grid_precision) zCoeffs[0],(grid_precision) zCoeffs[1]-gridDiff, (grid_precision) zCoeffs[2]};
        std::vector<grid_precision> end_point = {(grid_precision) xCoeffs[0], (grid_precision) xCoeffs[1]+gridDiff, (grid_precision) xCoeffs[2],
                                                 (grid_precision) yCoeffs[0], (grid_precision) yCoeffs[1]+gridDiff, (grid_precision) yCoeffs[2],
                                                 (grid_precision) zCoeffs[0], (grid_precision) zCoeffs[1]+gridDiff, (grid_precision) zCoeffs[2]};
        std::vector<grid_precision> grid_numSamples = {(grid_precision)1, (grid_precision) gridN, (grid_precision) 1,
                                                       (grid_precision)1, (grid_precision) gridN, (grid_precision) 1,
                                                       (grid_precision)1, (grid_precision) gridN, (grid_precision) 1};

        image_err_func_byvalue_quadratic host_func_byval_ptr;
        // Copy device function pointer for the function having by-value parameters to host side
        hipMemcpyFromSymbol(&host_func_byval_ptr, HIP_SYMBOL(dev_func_byvalue_ptr_quadratic),
                             sizeof(dev_func_byvalue_ptr_quadratic));

        for(int iii = 0; iii < multiRes; iii++) {
            CudaGrid<grid_precision, grid_dimension_quadratic> grid;
            ck(hipMalloc(&grid.data(), grid.bytesSize()));

            grid.setStartPoint(start_point);
            grid.setEndPoint(end_point);
            grid.setNumSamples(grid_numSamples);
            grid.display("grid");

            grid_precision axis_sample_counts[grid_dimension_quadratic];
            grid.getAxisSampleCounts(axis_sample_counts);

            CudaTensor<func_precision, grid_dimension_quadratic> func_values(axis_sample_counts);
            ck(hipMalloc(&func_values._data, func_values.bytesSize()));

            // first template argument is the error function return type
            // second template argument is the grid point value type
            CudaGridSearcher<func_precision, grid_precision, grid_dimension_quadratic> gridsearcher(grid, func_values);

            c1 = clock();
            gridsearcher.search_by_value_stream(host_func_byval_ptr, 1000, 451,
                    // gridsearcher.search_by_value(host_func_byval_ptr,
                                                data_p,
                                                numRSamples, numASamples,
                                                delta_x_m_per_pix, delta_y_m_per_pix,
                                                left_m, bottom_m,
                                                minRange, maxRange,
                                                ax_p,
                                                ay_p,
                                                az_p,
                                                sr_p,
                                                sf_p,
                                                sip_p,
                                                rv_p);
            c2 = clock();
            float searchTime = (float) (c2 - c1) * 1000 / CLOCKS_PER_SEC;
            printf("INFO: cuGridSearch took %f ms.\n", searchTime);

            totalTime += searchTime;

            func_precision min_value;
            int32_t min_value_index1d;
            func_values.find_extrema(min_value, min_value_index1d);

            grid_precision min_grid_point[grid_dimension_quadratic];
            grid.getGridPoint(min_grid_point, min_value_index1d);
            std::cout << "Minimum found at point p = { ";
            for (int d=0; d < grid_dimension_quadratic; d++) {
                minParams[d] = min_grid_point[d];
                std::cout << min_grid_point[d] << ((d < grid_dimension_quadratic - 1) ? ", " : " ");

                start_point[d] = min_grid_point[d] - (end_point[d] - start_point[d]) / 4;
                end_point[d] = min_grid_point[d] + (end_point[d] - start_point[d]) / 4;
                // Update grid here or outside (basically somewhere near here)
            }
            std::cout << "}" << std::endl;

            ck(hipFree(grid.data()));
            ck(hipFree(func_values.data()));

        }

        printf("Total time took: %f\n", totalTime);
        *myfile << "time," << totalTime << ',';

        printf("MinParams[");
        for(int d = 0; d < grid_dimension_quadratic; d++) {
            printf("%e,", minParams[d]);
        }
        printf("]\n");

        *myfile << "found,";
        for(int i = 0; i < grid_dimension_quadratic; i++)
            *myfile << minParams[i] << ',';

        nv_ext::Vec<grid_precision, grid_dimension_quadratic> minParamsVec(minParams);
        computeImageKernel<func_precision, grid_precision, grid_dimension_quadratic, __nTp><<<1,451>>>(minParamsVec,
                                                                                                       data_p,
                                                                                                       numRSamples, numASamples,
                                                                                                       delta_x_m_per_pix, delta_y_m_per_pix,
                                                                                                       left_m, bottom_m,
                                                                                                       minRange, maxRange,
                                                                                                       ax_p,
                                                                                                       ay_p,
                                                                                                       az_p,
                                                                                                       sr_p,
                                                                                                       sf_p,
                                                                                                       sip_p,
                                                                                                       rv_p,
                                                                                                       oi_p);

    }

//    grid_precision testHolder[] = {xCoeffs[0], xCoeffs[1], yCoeffs[0], yCoeffs[1], zCoeffs[0], zCoeffs[1]};
//    nv_ext::Vec<grid_precision, grid_dimension_quadratic> testHolderVec(testHolder);

    /* NOTE: COMMENT IF GRID ONLY */
    c1 = clock();
    printf("INFO: CUDA Backprojection kernel launch took %f ms.\n", (float) (c1 - c0) * 1000 / CLOCKS_PER_SEC);
    if (hipDeviceSynchronize() != hipSuccess)
        printf("\nERROR: threads did NOT synchronize! DO NOT TRUST RESULTS!\n\n");
    c2 = clock();
    printf("INFO: CUDA Backprojection execution took %f ms.\n", (float) (c2 - c1) * 1000 / CLOCKS_PER_SEC);
    printf("INFO: CUDA Backprojection total time took %f ms.\n", (float) (c2 - c0) * 1000 / CLOCKS_PER_SEC);
    /**/

    int num_img_bytes = sizeof (hipfftComplex) * sar_image_params.N_x_pix * sar_image_params.N_y_pix;
    std::vector<hipfftComplex> image_data(sar_image_params.N_x_pix * sar_image_params.N_y_pix);
    //cuda_res.copyFromDevice("output_image", &output_image[0], num_img_bytes);
    cuda_res.copyFromDevice("output_image", image_data.data(), num_img_bytes);
    for (int idx = 0; idx < sar_image_params.N_x_pix * sar_image_params.N_y_pix; idx++) {
        output_image[idx]._M_real = image_data[idx].x;
        output_image[idx]._M_imag = image_data[idx].y;
    }

    cuda_res.freeGPUMemory("range_vec");

    delete[] xCoeffs;
    delete[] yCoeffs;
    delete[] zCoeffs;

    if (finalize_CUDAResources(sar_data, sar_image_params, cuda_res) == EXIT_FAILURE) {
        std::cout << "cuda_focus_SAR_image::Problem found de-allocating and free resources on the GPU. Exiting..." << std::endl;
        return;
    }
    std::cout << cuda_res << std::endl;
}

int main(int argc, char **argv) {
    ComplexType test[] = {1.0, 1.0, 1.0, 1.0, 0.0, 0.0, 0.0, 0.0};
    ComplexType out[8];
    ComplexArrayType data(test, 8);
    std::unordered_map<std::string, matvar_t*> matlab_readvar_map;

    cxxopts::Options options("cpuBackProjection", "UNC Charlotte Machine Vision Lab SAR Back Projection focusing code.");
    cxxopts_integration(options);

    auto result = options.parse(argc, argv);

    if (result.count("help")) {
        std::cout << options.help() << std::endl;
        exit(0);
    }
    bool debug = result["debug"].as<bool>();
    int multiRes = result["multi"].as<int>();
    int style = result["style"].as<int>();
    int nPulse = result["numPulse"].as<int>();

    initialize_Sandia_SPHRead(matlab_readvar_map);
    initialize_GOTCHA_MATRead(matlab_readvar_map);

    std::string inputfile;
    if (result.count("input")) {
        inputfile = result["input"].as<std::string>();
    } else {
        std::stringstream ss;

        // Sandia SAR DATA FILE LOADING
        int file_idx = 9; // 1-10 for Sandia Rio Grande, 1-9 for Sandia Farms
        std::string fileprefix = Sandia_RioGrande_fileprefix;
        std::string filepostfix = Sandia_RioGrande_filepostfix;
        //        std::string fileprefix = Sandia_Farms_fileprefix;
        //        std::string filepostfix = Sandia_Farms_filepostfix;
        ss << std::setfill('0') << std::setw(2) << file_idx;


        // GOTCHA SAR DATA FILE LOADING
        int azimuth = 1; // 1-360 for all GOTCHA polarities=(HH,VV,HV,VH) and pass=[pass1,...,pass7] 
        //        std::string fileprefix = GOTCHA_fileprefix;
        //        std::string filepostfix = GOTCHA_filepostfix;
        //        ss << std::setfill('0') << std::setw(3) << azimuth;

        inputfile = fileprefix + ss.str() + filepostfix + ".mat";
    }

    std::cout << "Successfully opened MATLAB file " << inputfile << "." << std::endl;

    SAR_Aperture<NumericType> SAR_aperture_data;
    if (read_MAT_Variables(inputfile, matlab_readvar_map, SAR_aperture_data) == EXIT_FAILURE) {
        std::cout << "Could not read all desired MATLAB variables from " << inputfile << " exiting." << std::endl;
        return EXIT_FAILURE;
    }
    // Print out raw data imported from file
    std::cout << SAR_aperture_data << std::endl;

    // Sandia SAR data is multi-channel having up to 4 polarities
    // 1 = HH, 2 = HV, 3 = VH, 4 = VVbandwidth = 0:freq_per_sample:(numRangeSamples-1)*freq_per_sample;
    std::string polarity = result["polarity"].as<std::string>();
    if ((polarity == "HH" || polarity == "any") && SAR_aperture_data.sampleData.shape.size() >= 1) {
        SAR_aperture_data.polarity_channel = 0;
    } else if (polarity == "HV" && SAR_aperture_data.sampleData.shape.size() >= 2) {
        SAR_aperture_data.polarity_channel = 1;
    } else if (polarity == "VH" && SAR_aperture_data.sampleData.shape.size() >= 3) {
        SAR_aperture_data.polarity_channel = 2;
    } else if (polarity == "VV" && SAR_aperture_data.sampleData.shape.size() >= 4) {
        SAR_aperture_data.polarity_channel = 3;
    } else {
        std::cout << "Requested polarity channel " << polarity << " is not available." << std::endl;
        return EXIT_FAILURE;
    }
    if (SAR_aperture_data.sampleData.shape.size() > 2) {
        SAR_aperture_data.format_GOTCHA = false;
        // the dimensional index of the polarity index in the 
        // multi-dimensional array (for Sandia SPH SAR data)
        SAR_aperture_data.polarity_dimension = 2;
    }

    initialize_SAR_Aperture_Data(SAR_aperture_data);

    SAR_ImageFormationParameters<NumericType> SAR_image_params =
            SAR_ImageFormationParameters<NumericType>();

    // to increase the frequency samples to a power of 2
    // SAR_image_params.N_fft = (int) 0x01 << (int) (ceil(log2(SAR_aperture_data.numRangeSamples)));
    SAR_image_params.N_fft = (int)SAR_aperture_data.numRangeSamples;
    //SAR_image_params.N_fft = aperture.numRangeSamples;
    SAR_image_params.N_x_pix = (int)SAR_aperture_data.numAzimuthSamples;
    //SAR_image_params.N_y_pix = image_params.N_fft;
    SAR_image_params.N_y_pix = (int)SAR_aperture_data.numRangeSamples;
    // focus image on target phase center
    // Determine the maximum scene size of the image (m)
    // max down-range/fast-time/y-axis extent of image (m)
    SAR_image_params.max_Wy_m = CLIGHT / (2.0 * SAR_aperture_data.mean_deltaF);
    // max cross-range/fast-time/x-axis extent of image (m)
    SAR_image_params.max_Wx_m = CLIGHT / (2.0 * std::abs(SAR_aperture_data.mean_Ant_deltaAz) * SAR_aperture_data.mean_startF);

    // default view is 100% of the maximum possible view
    SAR_image_params.Wx_m = 1.00 * SAR_image_params.max_Wx_m;
    SAR_image_params.Wy_m = 1.00 * SAR_image_params.max_Wy_m;
    // make reconstructed image equal size in (x,y) dimensions
    SAR_image_params.N_x_pix = (int) ((float) SAR_image_params.Wx_m * SAR_image_params.N_y_pix) / SAR_image_params.Wy_m;
    // Determine the resolution of the image (m)
    SAR_image_params.slant_rangeResolution = CLIGHT / (2.0 * SAR_aperture_data.mean_bandwidth);
    SAR_image_params.ground_rangeResolution = SAR_image_params.slant_rangeResolution / std::sin(SAR_aperture_data.mean_Ant_El);
    SAR_image_params.azimuthResolution = CLIGHT / (2.0 * SAR_aperture_data.Ant_totalAz * SAR_aperture_data.mean_startF);

    // Print out data after critical data fields for SAR focusing have been computed
    std::cout << SAR_aperture_data << std::endl;

    SAR_Aperture<NumericType> SAR_focusing_data;
    if (!SAR_aperture_data.format_GOTCHA) {
        //SAR_aperture_data.exportData(SAR_focusing_data, SAR_aperture_data.polarity_channel);
        SAR_aperture_data.exportData(SAR_focusing_data, 2);
    } else {
        SAR_focusing_data = SAR_aperture_data;
    }

    //    SAR_ImageFormationParameters<NumericType> SAR_image_params =
    //            SAR_ImageFormationParameters<NumericType>::create<NumericType>(SAR_focusing_data);

    if(nPulse > 2) {
        SAR_focusing_data.numAzimuthSamples = nPulse;
    }

    std::cout << "Data for focusing" << std::endl;
    std::cout << SAR_focusing_data << std::endl;

    std::ofstream myfile;
    myfile.open("collectedData.txt", std::ios::out | std::ios::app);
    myfile << inputfile.c_str() << ',';

    printf("Main: deltaAz = %f, deltaF = %f, mean_startF = %f\nmaxWx_m = %f, maxWy_m = %f, Wx_m = %f, Wy_m = %f\nX_pix = %d, Y_pix = %d\nNum Az = %d, Num range = %d\n", SAR_aperture_data.mean_Ant_deltaAz, SAR_aperture_data.mean_startF, SAR_aperture_data.mean_deltaF, SAR_image_params.max_Wx_m, SAR_image_params.max_Wy_m, SAR_image_params.Wx_m, SAR_image_params.Wy_m, SAR_image_params.N_x_pix, SAR_image_params.N_y_pix, SAR_aperture_data.numAzimuthSamples, SAR_aperture_data.numRangeSamples);
    ComplexArrayType output_image(SAR_image_params.N_y_pix * SAR_image_params.N_x_pix);

    if (multiRes < 1) multiRes = 1;
    grid_cuda_focus_SAR_image(SAR_focusing_data, SAR_image_params, output_image, &myfile, multiRes, style);

    // Required parameters for output generation manually overridden by command line arguments
    std::string output_filename = result["output"].as<std::string>();
    SAR_image_params.dyn_range_dB = result["dynrange"].as<float>();

    writeBMPFile(SAR_image_params, output_image, output_filename);
    myfile << '\n';
    myfile.close();
    return EXIT_SUCCESS;
}
